#include "hip/hip_runtime.h"
#include "fluid_solver.h"
#include <cmath>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

#define IX(i, j, k) ((i) + (M + 2) * (j) + (M + 2) * (N + 2) * (k))
#define SWAP(x0, x)      \
    {                    \
        float *tmp = x0; \
        x0 = x;          \
        x = tmp;         \
    }
#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define LINEARSOLVERTIMES 20
#define BLOCK 8
#define BLOCK_SIZE BLOCK *BLOCK *BLOCK

__global__ void addSourceKernel(int M, int N, int O, float *x, float *s, float dt)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    if (idx < M + 2 && idy < N + 2 && idz < O + 2)
    {
        int index = IX(idx, idy, idz);
        x[index] += dt * s[index];
    }
}

void add_source(int M, int N, int O, float *x, float *s, float dt)
{
    dim3 block(BLOCK, BLOCK, BLOCK);
    dim3 grid((M + block.x - 1) / block.x,
              (N + block.y - 1) / block.y,
              (O + block.z - 1) / block.z);

    addSourceKernel<<<grid, block>>>(M, N, O, x, s, dt);
}

__global__ void set_bnd_kernel(int M, int N, int O, int b, float *x)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx <= M * N)
    {
        int i = idx % M + 1;
        int j = idx / M + 1;
        if (i <= M && j <= N)
        {
            x[IX(i, j, 0)] = b == 3 ? -x[IX(i, j, 1)] : x[IX(i, j, 1)];
            x[IX(i, j, O + 1)] = b == 3 ? -x[IX(i, j, O)] : x[IX(i, j, O)];
        }
    }

    if (idx <= M * O)
    {
        int i = idx % M + 1;
        int k = idx / M + 1;
        if (i <= M && k <= O)
        {
            x[IX(i, 0, k)] = b == 2 ? -x[IX(i, 1, k)] : x[IX(i, 1, k)];
            x[IX(i, N + 1, k)] = b == 2 ? -x[IX(i, N, k)] : x[IX(i, N, k)];
        }
    }

    if (idx <= N * O)
    {
        int j = idx % N + 1;
        int k = idx / N + 1;
        if (j <= N && k <= O)
        {
            x[IX(0, j, k)] = b == 1 ? -x[IX(1, j, k)] : x[IX(1, j, k)];
            x[IX(M + 1, j, k)] = b == 1 ? -x[IX(M, j, k)] : x[IX(M, j, k)];
        }
    }

    if (idx == 0) // Garantir que só uma thread faz isso
    {
        x[IX(0, 0, 0)] = 0.33f * (x[IX(1, 0, 0)] + x[IX(0, 1, 0)] + x[IX(0, 0, 1)]);
        x[IX(M + 1, 0, 0)] = 0.33f * (x[IX(M, 0, 0)] + x[IX(M + 1, 1, 0)] + x[IX(M + 1, 0, 1)]);
        x[IX(0, N + 1, 0)] = 0.33f * (x[IX(1, N + 1, 0)] + x[IX(0, N, 0)] + x[IX(0, N + 1, 1)]);
        x[IX(M + 1, N + 1, 0)] = 0.33f * (x[IX(M, N + 1, 0)] + x[IX(M + 1, N, 0)] + x[IX(M + 1, N + 1, 1)]);
        x[IX(0, 0, O + 1)] = 0.33f * (x[IX(1, 0, O + 1)] + x[IX(0, 1, O + 1)] + x[IX(0, 0, O)]);
        x[IX(M + 1, 0, O + 1)] = 0.33f * (x[IX(M, 0, O + 1)] + x[IX(M + 1, 1, O + 1)] + x[IX(M + 1, 0, O)]);
        x[IX(0, N + 1, O + 1)] = 0.33f * (x[IX(1, N + 1, O + 1)] + x[IX(0, N, O + 1)] + x[IX(0, N + 1, O)]);
        x[IX(M + 1, N + 1, O + 1)] = 0.33f * (x[IX(M, N + 1, O + 1)] + x[IX(M + 1, N, O + 1)] + x[IX(M + 1, N + 1, O)]);
    }
}

void set_bnd(int M, int N, int O, int b, float *x)
{
    int maxSize = MAX(MAX(M * N, M * O), N * O);
    int block = BLOCK_SIZE;
    int numBlocks = (maxSize + block - 1) / block;
    set_bnd_kernel<<<numBlocks, block>>>(M, N, O, b, x);
}

__global__ void lin_solve_kernel(int M, int N, int O, float *x, float *x0,
                                float a, float c, int phase, float *max_change)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int j = blockIdx.y * (blockDim.y - 2) + ty;
    int k = blockIdx.z * (blockDim.z - 2) + tz;
    int i = 2 * (blockIdx.x * (blockDim.x - 2) + tx) + ((j + k) % 2 == phase ? 0 : 1); 
    if (tx > 0 && tx < blockDim.x -1 &&
        ty > 0 && ty < blockDim.y -1 &&
        tz > 0 && tz < blockDim.z -1 &&
        i <= M && j <= N && k <= O)
    {   
        float old_x = x[IX(i, j, k)];
        float new_x = (x0[IX(i, j, k)] +
                       a * (x[IX(i-1, j, k)] + x[IX(i+1, j, k)] +
                            x[IX(i, j-1, k)] + x[IX(i, j+1, k)] +
                            x[IX(i, j, k-1)] + x[IX(i, j, k+1)])) / c;

        x[IX(i, j, k)] = new_x;
        float change = fabsf(new_x - old_x);
        if (change > *max_change)
        {
            atomicMax((int *)max_change, __float_as_int(change));
        }
    }
}
void lin_solve(int M, int N, int O, int b, float *x, float *x0, float a, float c)
{
    dim3 block(BLOCK - 1, BLOCK - 1, BLOCK - 1);
    dim3 grid(((M/2) + block.x - 1) / block.x,
              (N + block.y - 1) / block.y,
              (O + block.z - 1) / block.z);
    float *max_change_dev;
    hipMalloc(&max_change_dev, sizeof(float));
    float max_change_host;
    int iter = 0;
    float tol = 1e-7f;
    do
    {
        float init_max = 0.0f;
        hipMemcpy(max_change_dev, &init_max, sizeof(float), hipMemcpyHostToDevice);
        lin_solve_kernel<<<grid, block>>>(M, N, O, x, x0, a, c, 0, max_change_dev);
        lin_solve_kernel<<<grid, block>>>(M, N, O, x, x0, a, c, 1, max_change_dev);
        set_bnd_kernel<<<(MAX(MAX(M, N), O) + 255) / 256, 256>>>(M, N, O, b, x);
        hipMemcpy(&max_change_host, max_change_dev, sizeof(float), hipMemcpyDeviceToHost);
        iter++;
    } while (max_change_host > tol && iter < LINEARSOLVERTIMES);
    hipFree(max_change_dev);
}

void diffuse(int M, int N, int O, int b, float *x, float *x0, float diff,
             float dt)
{
    int max = MAX(MAX(M, N), O);
    float a = dt * diff * max * max;
    lin_solve(M, N, O, b, x, x0, a, 1 + 6 * a);
}

__global__ void advectKernel(int M, int N, int O, int b, float *d, float *d0,
                             float *u, float *v, float *w, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O)
    {
        float dtX = dt * M;
        float dtY = dt * N;
        float dtZ = dt * O;

        float x = i - dtX * u[IX(i, j, k)];
        float y = j - dtY * v[IX(i, j, k)];
        float z = k - dtZ * w[IX(i, j, k)];

        x = fminf(fmaxf(x, 0.5f), M + 0.5f);
        y = fminf(fmaxf(y, 0.5f), N + 0.5f);
        z = fminf(fmaxf(z, 0.5f), O + 0.5f);

        int i0 = (int)x;
        int i1 = i0 + 1;
        int j0 = (int)y;
        int j1 = j0 + 1;
        int k0 = (int)z;
        int k1 = k0 + 1;

        float s1 = x - i0;
        float s0 = 1.0f - s1;
        float t1 = y - j0;
        float t0 = 1.0f - t1;
        float u1 = z - k0;
        float u0 = 1.0f - u1;

        d[IX(i, j, k)] =
            s0 * (t0 * (u0 * d0[IX(i0, j0, k0)] + u1 * d0[IX(i0, j0, k1)]) +
                  t1 * (u0 * d0[IX(i0, j1, k0)] + u1 * d0[IX(i0, j1, k1)])) +
            s1 * (t0 * (u0 * d0[IX(i1, j0, k0)] + u1 * d0[IX(i1, j0, k1)]) +
                  t1 * (u0 * d0[IX(i1, j1, k0)] + u1 * d0[IX(i1, j1, k1)]));
    }
}

void advect(int M, int N, int O, int b, float *d, float *d0,
            float *u, float *v, float *w, float dt)
{
    dim3 block(BLOCK, BLOCK, BLOCK);
    dim3 grid(
        (M + block.x - 1) / block.x,
        (N + block.y - 1) / block.y,
        (O + block.z - 1) / block.z);

    advectKernel<<<grid, block>>>(M, N, O, b, d, d0, u, v, w, dt);

    set_bnd(M, N, O, b, d);
}

__global__ void project_kernel_1(int M, int N, int O, float *u, float *v,
                                 float *w, float *p, float *div, float max1)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O)
    {
        div[IX(i, j, k)] = -0.5f *
                           (u[IX(i + 1, j, k)] - u[IX(i - 1, j, k)] +
                            v[IX(i, j + 1, k)] - v[IX(i, j - 1, k)] +
                            w[IX(i, j, k + 1)] - w[IX(i, j, k - 1)]) /
                           max1;

        p[IX(i, j, k)] = 0.0f;
    }
}

__global__ void project_kernel_2(int M, int N, int O, float *u, float *v,
                                 float *w, float *p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O)
    {
        u[IX(i, j, k)] -= 0.5f * (p[IX(i + 1, j, k)] - p[IX(i - 1, j, k)]);
        v[IX(i, j, k)] -= 0.5f * (p[IX(i, j + 1, k)] - p[IX(i, j - 1, k)]);
        w[IX(i, j, k)] -= 0.5f * (p[IX(i, j, k + 1)] - p[IX(i, j, k - 1)]);
    }
}

void project(int M, int N, int O, float *u, float *v, float *w, float *p, float *div)
{
    float max1 = (float)MAX(M, MAX(N, O));

    dim3 block(BLOCK, BLOCK, BLOCK);
    dim3 grid(
        (M + block.x - 1) / block.x,
        (N + block.y - 1) / block.y,
        (O + block.z - 1) / block.z);

    project_kernel_1<<<grid, block>>>(M, N, O, u, v, w, p, div, max1);

    set_bnd(M, N, O, 0, div);
    set_bnd(M, N, O, 0, p);

    lin_solve(M, N, O, 0, p, div, 1, 6);

    project_kernel_2<<<grid, block>>>(M, N, O, u, v, w, p);

    set_bnd(M, N, O, 1, u);
    set_bnd(M, N, O, 2, v);
    set_bnd(M, N, O, 3, w);
}


void dens_step(int M, int N, int O, float *x, float *x0, float *u, float *v,
               float *w, float diff, float dt)
{
    add_source(M, N, O, x, x0, dt);
    // SWAP(x0, x);
    diffuse(M, N, O, 0, x0, x, diff, dt);
    // SWAP(x0, x);
    advect(M, N, O, 0, x, x0, u, v, w, dt);
}

void vel_step(int M, int N, int O, float *u, float *v, float *w, float *u0,
              float *v0, float *w0, float visc, float dt)
{
    add_source(M, N, O, u, u0, dt);
    add_source(M, N, O, v, v0, dt);
    add_source(M, N, O, w, w0, dt);
    // SWAP(u0, u);
    diffuse(M, N, O, 1, u0, u, visc, dt); // SWAP(u0, u);
    // SWAP(v0, v);
    diffuse(M, N, O, 2, v0, v, visc, dt); // SWAP(v0, v);
    // SWAP(w0, w);
    diffuse(M, N, O, 3, w0, w, visc, dt); // SWAP(w0, w);
    project(M, N, O, u0, v0, w0, u, v);   // 3 swaps ao mesmo tempo

    // SWAP(u0, u);
    // SWAP(v0, v);
    // SWAP(w0, w);

    // Aqui voltamos aos originais pois 2 swaps é o mesmo que nada
    advect(M, N, O, 1, u, u0, u0, v0, w0, dt);
    advect(M, N, O, 2, v, v0, u0, v0, w0, dt);
    advect(M, N, O, 3, w, w0, u0, v0, w0, dt);
    project(M, N, O, u, v, w, u0, v0);
}
