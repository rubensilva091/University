#include "hip/hip_runtime.h"
#include "EventManager.h"
#include "fluid_solver.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>
#include <time.h>

#define SIZE 168

#define IX(i, j, k) ((i) + (M + 2) * (j) + (M + 2) * (N + 2) * (k))


static int M = SIZE;
static int N = SIZE;
static int O = SIZE;
static float dt = 0.1f;     
static float diff = 0.0001f; 
static float visc = 0.0001f; 


//Host
static float *u, *v, *w, *u_prev, *v_prev, *w_prev;
static float *dens, *dens_prev;
//Device
static float *d_u, *d_v, *d_w, *d_u_prev, *d_v_prev, *d_w_prev;
static float *d_dens, *d_dens_prev;



int allocate_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  u = new float[size];
  v = new float[size];
  w = new float[size];
  u_prev = new float[size];
  v_prev = new float[size];
  w_prev = new float[size];
  dens = new float[size];
  dens_prev = new float[size];

  if (!u || !v || !w || !u_prev || !v_prev || !w_prev || !dens || !dens_prev) {
    std::cerr << "Cannot allocate memory" << std::endl;
    return 0;
  }
  return 1;
}

void allocate_cuda()
{
  int size = (M + 2) * (N + 2) * (O + 2);
  hipMalloc(&d_u, size * sizeof(float));
  hipMalloc(&d_v, size * sizeof(float));
  hipMalloc(&d_w, size * sizeof(float));
  hipMalloc(&d_u_prev, size * sizeof(float));
  hipMalloc(&d_v_prev, size * sizeof(float));
  hipMalloc(&d_w_prev, size * sizeof(float));
  hipMalloc(&d_dens, size * sizeof(float));
  hipMalloc(&d_dens_prev, size * sizeof(float));
}

void memcpy_host_to_device()
{
  int size = (M + 2) * (N + 2) * (O + 2);
  hipMemcpy(d_u, u, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v, v, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w, w, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_u_prev, u_prev, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v_prev, v_prev, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w_prev, w_prev, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dens, dens, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dens_prev, dens_prev, size * sizeof(float), hipMemcpyHostToDevice);
}

void memcpy_device_to_host()
{
  int size = (M + 2) * (N + 2) * (O + 2);
  hipMemcpy(u, d_u, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(v, d_v, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(w, d_w, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(u_prev, d_u_prev, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(v_prev, d_v_prev, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(w_prev, d_w_prev, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(dens, d_dens, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(dens_prev, d_dens_prev, size * sizeof(float), hipMemcpyDeviceToHost);
}

void free_cuda()
{
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
  hipFree(d_u_prev);
  hipFree(d_v_prev);
  hipFree(d_w_prev);
  hipFree(d_dens);
  hipFree(d_dens_prev);
}


void clear_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    u[i] = v[i] = w[i] = u_prev[i] = v_prev[i] = w_prev[i] = dens[i] =
        dens_prev[i] = 0;
  }
}

void free_data() {
  delete[] u;
  delete[] v;
  delete[] w;
  delete[] u_prev;
  delete[] v_prev;
  delete[] w_prev;
  delete[] dens;
  delete[] dens_prev;
}


__global__ void apply_events_kernel(float* d_dens, float* d_u, float* d_v, float* d_w, 
                                  int i, int j, int k, float density, float fx, float fy, float fz, 
                                  int event_type, int M, int N, int O) {
    int index = IX(i, j, k);
    if (event_type == ADD_SOURCE) {
        d_dens[index] = density;
    } 
    else if (event_type == APPLY_FORCE) {
        d_u[index] = fx;
        d_v[index] = fy;
        d_w[index] = fz;
    }
}


void apply_events(const std::vector<Event>& events) {
    for (const auto& event : events) {
       
        int i = M / 2, j = N / 2, k = O / 2;
        
        apply_events_kernel<<<1, 1>>>(d_dens, d_u, d_v, d_w,
                                    i, j, k,
                                    event.density,
                                    event.force.x, event.force.y, event.force.z,
                                    event.type,
                                    M, N, O);
    }
}

void simulate(EventManager& eventManager, int timesteps) {
    memcpy_host_to_device(); 

    for (int t = 0; t < timesteps; t++) {
        
        std::vector<Event> events = eventManager.get_events_at_timestamp(t);
        apply_events(events);

        vel_step(M, N, O, d_u, d_v, d_w, d_u_prev, d_v_prev, d_w_prev, visc, dt);
        dens_step(M, N, O, d_dens, d_dens_prev, d_u, d_v, d_w, diff, dt);
    }
    
    memcpy_device_to_host();
}

float sum_density() {
  float total_density = 0.0f;
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    total_density += dens[i];
  }
  return total_density;
}

int main() {
  EventManager eventManager;
  eventManager.read_events("src/events.txt");

  int timesteps = eventManager.get_total_timesteps();

  if (!allocate_data())
    return -1;
  clear_data();
  allocate_cuda();


  clock_t start = clock();

  simulate(eventManager, timesteps);


  clock_t end = clock();
  std::cout << "Simulation took CPU " << (float)(end - start) / CLOCKS_PER_SEC << " seconds." << std::endl;
  float total_density = sum_density();
  std::cout << "Total density after " << timesteps
            << " timesteps: " << total_density << std::endl;

  free_data(); //Host
  free_cuda(); //Device

  return 0;
}
